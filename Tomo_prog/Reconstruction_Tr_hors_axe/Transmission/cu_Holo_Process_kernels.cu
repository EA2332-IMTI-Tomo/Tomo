#include "hip/hip_runtime.h"
#include "cu_Holo_Process_kernels.h"



// *****************************************************************************
// 3D mapping 
// *****************************************************************************


// -----------------------------------------------------------------------------
// kernel CUDA


__global__ void
kernel_mapping_float(int l_xm0, int l_ym0, size_t p_Nxmax, size_t p_Nymax, \
		     float p_rayon, float p_delta_zmax, double zm0,	\
		     size_t c_dv0s2rf, size_t c_dv1s2rf, size_t c_dv0rf, size_t c_dv2s2rf, size_t c_dv1xdv0rf, \
		     unsigned short int *sup_redon_d, float *reel_arc_d, float *imag_arc_d, \
		     double *fft_reel_shift_norm_d, double *fft_imag_shift_norm_d)
{
  int s = CUDAINDEXE;
  const size_t tabsize = 4 * p_Nxmax * p_Nymax;

  if (s < tabsize)
    {
      const size_t c_Nxmax2 = 2 * p_Nxmax, c_Nymax2 = 2 * p_Nymax;
      const float c_rayon_sqr = p_rayon * p_rayon;
      const float c_delta_zmax_sqr = p_delta_zmax * p_delta_zmax;
      const int c_scan_size = 2 * p_Nxmax * 2 * p_Nymax;

      double arg_z_arc, z_arc; 
      int k, i, j, _x, _y;
      
      i = s / c_Nxmax2;
      j = s % c_Nxmax2;
      _y = i - p_Nxmax;
      _x = j - p_Nymax;
    
      //argument sous la racine calcul dans ARC_centre
      arg_z_arc = c_rayon_sqr - _x*_x - _y*_y;

      //ne pas depasser l'ouverture numérique pour 1 hologramme
      if(arg_z_arc > c_delta_zmax_sqr)
	{
	  //altitude au centre du volume
	  z_arc = round(sqrt(arg_z_arc) - zm0);
	  
	  //indice du tableau 1D du volume 3D
	  k = round(( - l_xm0 + _x + c_dv0s2rf) + (- l_ym0 + _y + c_dv1s2rf) * c_dv0rf + \
		    (z_arc + c_dv2s2rf) * c_dv1xdv0rf);

	  sup_redon_d[k] += 1;//pour calculer le support
	  
	  reel_arc_d[k] += fft_reel_shift_norm_d[s];//pour calculer l'image
	  imag_arc_d[k] += fft_imag_shift_norm_d[s];//pour calculer l'image
	}
    }
}





// *****************************************************************************
// centered 2D crop
// *****************************************************************************



// -----------------------------------------------------------------------------
// kernel CUDA


__global__ void
kernel_crop2D_double(double *src_data, double *dst_data, size_t src_size, size_t dst_size)
{
  int s = CUDAINDEXE;
  size_t tabsize = src_size * src_size;

  size_t xs, ys, xd, yd;
  size_t xmin, xmax;
  size_t delta;

  //ASSERT( src_size >= dst_size);

  if (s < tabsize)
    {
      delta = (src_size - dst_size) / 2;

      ys = s / src_size;
      xs = s % src_size;

      xd = xs - delta;
      yd = ys - delta;

      if (xd >= 0 && xd < dst_size && \\
	  yd >= 0 && yd < dst_size)
	dst_data[ yd * dst_size + xd ] = src_data[ ys * src_size + xs ];
    }
    
}



// *****************************************************************************
// peak_normalize
// *****************************************************************************


// 	const size_t cpt_max = peak_normalize(c_NxNy4, fft_reel_shift_h, fft_imag_shift_h, \
// 					      fft_reel_shift_norm, fft_imag_shift_norm);

// -----------------------------------------------------------------------------
// kernel CUDA



__global__ void
kernel_norm_double(double *dst_data, double *src_data_x, double *src_data_y, size_t src_size)
{
  int s = CUDAINDEXE;
  if (s < src_size)
    {
      double X = src_data_x[s];      
      double Y = src_data_y[s];
      dst_data[s] = X * X + Y * Y; 
    }
}


__global__ void
kernel_normpeak_double(double *dst_data_r, double *dst_data_i, double *src_data_r, double *src_data_i, size_t src_size, double max_part_reel, double max_part_imag, double inv_max_mod)
{
  int s = CUDAINDEXE;
  if (s < src_size)
    {
      double REAL = src_data_r[s];
      double IMAG = src_data_i[s];
      dst_data_r[s] = (REAL * max_part_reel + IMAG * max_part_imag) * inv_max_mod;
      dst_data_i[s] = (IMAG * max_part_reel - REAL * max_part_imag) * inv_max_mod;
    }
}


// data_r/i : données e/s. 
// data_size_edge: côté du carré que le tableau r/i représente
// data_mask: tableau contenant le masque à appliquer
// data_mask_edge: on suppose le masque carré, on donne uniquement la valeur du côté
// spot: on applique le masque aux images r et i centré sur spot
__global__ void
kernel_applymask_double(double* data_r, double* data_i, size_t data_size_edge, \
			unsigned char* data_mask, size_t data_mask_edge, \
			size_t spot_x, size_t spot_y)
{
  const size_t data_mask_halfedge = data_mask_edge / 2;
  const size_t x_min = spot_x - data_mask_halfedge;
  const size_t x_max = spot_x + data_mask_halfedge;
  const size_t y_min = spot_y - data_mask_halfedge;
  const size_t y_max = spot_y + data_mask_halfedge;
  const double inv_255 = 1.0f / 255.0f;
  const size_t data_size = data_size_edge * data_size_edge;

  int s = CUDAINDEXE;
 
  if (s < data_size)
    {
      size_t x = s / data_size_edge;
      size_t y = s % data_size_edge;

      // si le voxel est dans la fenêtre
      if (( x > x_min ) && ( x < x_max ) && ( y > y_min ) && ( y < y_max ))
	{
	  size_t twin_index = data_mask_edge * ( y - spot_y + data_mask_halfedge ) + \
	    x - spot_x + data_mask_halfedge;
	  double twin_val_inv255 = data_mask[ twin_index ] * inv_255;
	  
	  data_r[s] *= twin_val_inv255;
	  data_i[s] *= twin_val_inv255;
	}
    }
}
