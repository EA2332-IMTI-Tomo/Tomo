// a mutex object
pthread_mutex_t mutexCUDAInit;

//----------------------------------------------------------------------------------------------------------
void *run_func(void *arg)
{
  // calling hipFree(0) within a critical section protected by the mutex
  pthread_mutex_lock(&mutexCUDAInit);
  hipFree(0);
  pthread_mutex_unlock(&mutexCUDAInit);

  // your CUDA code here

  return NULL;
}

#define NUM_THREAD	10
//----------------------------------------------------------------------------------------------------------
int main(int argc, char *argv[])
{
  pthread_t pt[NUM_THREAD];
  int i;
  int status;

  // initializing a mutex
  pthread_mutex_init(&mutexCUDAInit, NULL);

  // create threads
  for(i = 0 ; i < NUM_THREAD ; i++){
    printf("create thread %d\n", i);
    status = pthread_create(&pt[i], NULL, run_func, NULL);
    if (status != 0) {
      perror("pthread_create");
      return 2;
    }
  }

  for(i = 0 ; i < NUM_THREAD ; i++){
    pthread_join(pt[i], NULL);
  }

  return 0;
}


