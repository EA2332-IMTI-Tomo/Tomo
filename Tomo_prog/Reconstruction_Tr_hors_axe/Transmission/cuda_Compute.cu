#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <stdlib.h>

#include "macros.h"
#include "vChronos.h" // requiert boost

#include <boost/thread.hpp>  
#include <boost/date_time.hpp>

// -------------------------------
#include "main.h"
#include "util.h"
//#include "Holo_Process.h"
#include "cu_Holo_Process.h"

// -------------------------------
#include "cu_DisplayVolume.h"
#include "cpu_COMPLEXVolume.h"
#include "cu_ComplexVolume.h"
#include "cu_Volume.h"
#include "cuFFT_Volume.h"

// -------------------------------
using namespace std;

#include "cv.h"
#include "highgui.h"

#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
using namespace cv;



//******************************************************************************
// 
//******************************************************************************


extern char *g_OUTPUT_DIR;
extern char *g_OUTPUT_RADIX;
extern size_t g_fftw_threads;
//extern bool g_save_sup_redon; 

extern size_t g_window_dim_x;
extern size_t g_window_dim_y;


//==============================================================================
// Thread de visualisation de volume
//==============================================================================


void 
BoostThreadFuncGPU(cu_DisplayVolume<RECON_TYPE> *slicer)
{
  char c;

  while(true)
    {
      slicer -> updateImage();
      slicer -> showImage();
      c = cvWaitKey(15);
    }
  
         
  cout << "Worker: finished" << endl;
}


// =============================================================================
//  traite les batches d'hologrammes sur CPU, envoie sup_redon et le volume des fréquences sur le GPU pour calculer la reconstruction courante
//
//  La visualisation s'effectue à partir du GPU
//
// =============================================================================


void
compute_gpu_online_batch(size_t cube_edge, size_t Nxmax, size_t Nymax, size_t Nxmax_Rf, \
			 size_t window_edge_x, size_t window_edge_y, size_t image_dim_x, size_t image_dim_y, \
			 size_t xm0_limite, size_t ym0_limite, float rayon, float delta_zmax, \
			 size_t batch_size, \
			 const string &images_radix)
{

  // ==========================================================
  // Allocation des volumes de données
  // ==========================================================


  // CPU: volume des fréquences de fourier, rempli par la boucle de lecture
  cpu_COMPLEXVolume<RECON_TYPE> VC_Fourier_h(cube_edge, cube_edge, cube_edge);
  VC_Fourier_h.allocate();
  // pointeurs sur les sous-volumes
  AIR_Volume<RECON_TYPE>* V_FourierR_h = VC_Fourier_h.get_real_vol();
  AIR_Volume<RECON_TYPE>* V_FourierI_h = VC_Fourier_h.get_imag_vol();


  // GPU: le même
  cu_ComplexVolume<RECON_TYPE> VC_Fourier_d(cube_edge, cube_edge, cube_edge);
  VC_Fourier_d.allocate();
  VC_Fourier_d.fill(0);
  cu_Volume<RECON_TYPE>* V_FourierR_d = VC_Fourier_d.get_real_vol();
  cu_Volume<RECON_TYPE>* V_FourierI_d = VC_Fourier_d.get_imag_vol();


  // GPU: volume reconstruit à l'itération i, côté GPU
  cu_ComplexVolume<RECON_TYPE> VC_Visu_d(cube_edge, cube_edge, cube_edge);
  VC_Visu_d.allocate();
  VC_Visu_d.fill(0); // pour éviter le ghosting de la dernière acquisition
    // pointeurs sur les sous-volumes
  cu_Volume<RECON_TYPE>* V_VisuR_d = VC_Visu_d.get_real_vol();
  cu_Volume<RECON_TYPE>* V_VisuI_d = VC_Visu_d.get_imag_vol();



  // CPU: à la fin du calcul, volume qui va réceptionner la dernière reconstruction et la sauver sur disque
  cpu_COMPLEXVolume<RECON_TYPE> VC_Svg_h(cube_edge, cube_edge, cube_edge);
  VC_Svg_h.allocate();
  

  // GPU: on crée un volume calculable au format cuFFT
  cuFFT_Volume VF_Fourier_d(cube_edge, cube_edge, cube_edge);
  VF_Fourier_d.allocate();
  VF_Fourier_d.fill(0, 0);
  

  // CPU: on crée sup_redon pour normalisation dans fourier
  cpu_AIRVolume<unsigned short int> V_SupRedon_h(cube_edge, cube_edge, cube_edge);
  V_SupRedon_h.set_data_linear_mode(true);
  V_SupRedon_h.allocate();
  
  

  // GPU: le même
  cu_Volume<unsigned short int> V_SupRedon_d(cube_edge, cube_edge, cube_edge);
  V_SupRedon_d.allocate();
  V_SupRedon_d.fill(0);
  cu_Volume<unsigned short int> V_SupRedon2_d(cube_edge, cube_edge, cube_edge);
  V_SupRedon2_d.allocate();
  V_SupRedon2_d.fill(0);

  // ==========================================================
  // Début du Programme effectif
  // ==========================================================
    

  birdy();
  
  
  // ---------------------------------------------------
  // PRÉPARATION visu
  ASSERT(g_window_dim_x == g_window_dim_y);
  cu_DisplayVolume<RECON_TYPE> cu_VolSlicer("kilébo", V_VisuI_d);
  

  

  // *******************************************************
  // lecture des hologrammes sur CPU

  
  cu_Holo_Process Hologram_Processor;
  // Hologram_Processor.set_output_volumes(V_FourierR_h, V_FourierI_h, &V_SupRedon_h); 

  Hologram_Processor.set_output_volumes_gpu(V_FourierR_d -> get_data(), V_FourierI_d -> get_data(), V_SupRedon_d.get_data()); 
  
  Hologram_Processor.set_image_settings(image_dim_x, image_dim_y, window_edge_x, window_edge_y, g_window_dim_x);
  Hologram_Processor.set_rayon_settings(Nxmax, Nymax, Nxmax_Rf, xm0_limite, ym0_limite, rayon, delta_zmax);
  Hologram_Processor.set_file_settings(images_radix.c_str());
  Hologram_Processor.set_prepare();

  Hologram_Processor.display();



  size_t max_batch_error = 2; // was 2
  size_t errors = 0;
  size_t batch_num = 0;
  bool break_batches = false;
  char c;



  // ===========================================================================
  // LANCEMENT DU THREAD GUI en // 
  boost::thread workerThread(BoostThreadFuncGPU, &cu_VolSlicer);
  


  // ===========================================================================
  // BOUCLE DE TRAITEMENT DES BATCHES
  while(! break_batches)
    {

      // --------------------------------------------------
      // création d'un batch temps-réel (seuil max au-dela duquel on traite de force
      //, taille mini sous laquelle on attend, intervalle de dodo
      while (! Hologram_Processor.batch_prepare_realtime(400, 3, 50))
	{
	  errors++;
	  if (errors > max_batch_error)
	    {
	      cerr << endl << "recon:: probably reached end of acquisition, halting"; 
	      cerr.flush();
	      break_batches = true;
	      break;
	    }
	}


      
      // only exit point possible
      if (break_batches) 
	{
	  //vCHRONO_STOP(processing_time);
	  cerr << endl << "recon:: out with it";
	  
	  cout << endl << "recon:: waiting for keypress to exit";
	  cin.get(); //c = cvWaitKey(0); //
	  break;
	}

      
      // --------------------------------------------------
      // si la création a réussi

      batch_num++;
      size_t _batch_size = Hologram_Processor.batch_query_size();
      size_t _batch_start = Hologram_Processor.batch_query_angle_last_done() + 1;
      cout << endl << "batch n°" << batch_num << " [" <<  _batch_start\
	   << "; " << _batch_start - 1 + _batch_size << "]  ||" << _batch_size << "||";
      cout.flush();

      errors = 0;

      // batch launch!
      Hologram_Processor.batch_launch();

      
      // ==========================================================
      // Exécution de FFT 3D précédée et suivie de shifts circulaires (2.5s cpu)
      // ==========================================================


      // on a donné les volumes de VC_Fourier_d à Holo_Compute (gpu)
      //VF_Fourier_d.import_from( VC_Fourier_d );
      VF_Fourier_d.import_from_andfriend( VC_Fourier_d, V_SupRedon_d, V_SupRedon2_d );

      // et aussi V_SupRedon_d
      //V_SupRedon_d.circshift_to( V_SupRedon2_d );
      V_SupRedon2_d.nozero(1);
      
      VF_Fourier_d /= V_SupRedon2_d;

      VF_Fourier_d.set_fourier_backward();

      VC_Visu_d.fill(0);
      VF_Fourier_d.export_to( VC_Visu_d );

      cu_VolSlicer.notify_data_changed();      
      //hipDeviceSynchronize();  


    }
  // FIN BOUCLE BATCHES
  // ===========================================================================


  // *******************************************************
  // rapatriement du dernier volume reconstruit en GPU vers CPU
  VC_Visu_d.push_to_cpu( VC_Svg_h );
  

  // *******************************************************
  // sauvegardes sur disque

  bool g_save_sup_redon = false;
  
  

  string s_file_radix(g_OUTPUT_DIR); s_file_radix = s_file_radix +  "/" + g_OUTPUT_RADIX; 
  VC_Svg_h.change_files(s_file_radix);
  VC_Svg_h.write_files();
   
  
  if (g_save_sup_redon)
    {
      cerr << "unsupported";
      /*
      string s_filename(g_OUTPUT_DIR); s_filename += OUTPUT_REDON_FILENAME;
      V_SupRedon_h.change_files(s_filename.c_str());
      V_SupRedon_h.write_files();
      */
    }
      
  // *******************************************************
  // fin
  
  


}





      /*
      // ceci fonctionne très bien: Sup_Redon et Fréquences sur le CPU

      // send supredon to GPU
      V_SupRedon_d.pull_from_cpu( V_SupRedon_h );
      V_SupRedon_d.copy_to( V_SupRedon2_d );
      V_SupRedon2_d.nozero(1);

      // also send frequencies volume
      VC_Fourier_d.pull_from_cpu( VC_Fourier_h );
      

      // normalisation
      VC_Fourier_d /= V_SupRedon2_d;


      // and circshift and convert to cuFFT
      VF_Fourier_d.import_from( VC_Fourier_d );

      VF_Fourier_d.set_fourier_backward();
      
      // circshift et conversion inverse
      // mais copie sur le volume GPU de visu
      VF_Fourier_d.export_to( VC_Visu_d );
      cu_VolSlicer.notify_data_changed();

      */
